#include "hip/hip_runtime.h"
#include "DAGCompression/DAGCompression.h"
#include "GpuPrimitives.h"

// Returns NumWords
uint32 ComputeChildPositions(const FGpuLevel& Level, TStaticArray<uint32, EMemoryType::GPU>& ChildPositions)
{
	const int32 Num = Cast<int32>(Level.ChildMasks.Num());
	check(Num > 0);

	ChildPositions = TStaticArray<uint32, EMemoryType::GPU>("ChildPositions", Num);

	const auto NodeSizes = thrust::make_transform_iterator(Level.ChildMasks.GetData(),  [] GPU_LAMBDA (uint8 ChildMask) { return Utils::TotalSize(ChildMask); });

	{
		void* TempStorage = nullptr;
		size_t TempStorageBytes;
		CUDA_CHECKED_CALL hipcub::DeviceScan::ExclusiveSum(TempStorage, TempStorageBytes, NodeSizes, ChildPositions.GetData(), Num);
		CUDA_CHECKED_CALL cnmemMalloc(&TempStorage, TempStorageBytes, DEFAULT_STREAM);
		CUDA_CHECKED_CALL hipcub::DeviceScan::ExclusiveSum(TempStorage, TempStorageBytes, NodeSizes, ChildPositions.GetData(), Num);
		CUDA_CHECK_ERROR();
		CUDA_CHECKED_CALL cnmemFree(TempStorage, DEFAULT_STREAM);
	}
	
	return GetElement(ChildPositions, Num - 1) + Utils::TotalSize(GetElement(Level.ChildMasks, Num - 1));
}

template<typename T>
void WriteLevelTo(const FGpuLevel& Level, const TStaticArray<uint32, EMemoryType::GPU>& ChildPositions, TStaticArray<uint32, EMemoryType::GPU>& Data, T GetChildIndex)
{
	DAGCompression::CheckLevelIndices(Level);
	
	Scatter(Level.ChildMasks, ChildPositions, Data);

	const auto ChildrenIndices = Level.ChildrenIndices.CastTo<uint32>();
	const auto ChildMasks = Level.ChildMasks;

	ScatterIfWithTransform(
		ChildrenIndices, [=] GPU_LAMBDA(uint32 Index) { return Index == 0xFFFFFFFF ? Index : GetChildIndex(Index); },
		[=] GPU_LAMBDA (uint64 Index)
		{
			return ChildPositions[Index / 8] + Utils::ChildOffset(ChildMasks[Index / 8], Index % 8);
		},
		Data,
		[=] GPU_LAMBDA (uint64 Index)
		{
			checkf((ChildrenIndices[Index] == 0xFFFFFFFF) == !(ChildMasks[Index / 8] & (1 << (Index % 8))), "Index: %" PRIu64, Index);
			return ChildrenIndices[Index] != 0xFFFFFFFF;
		});
}

///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////

template<uint32 Level>
HOST_DEVICE_RECURSIVE uint64 SetLeavesCounts(
	TStaticArray<uint64, EMemoryType::GPU> Counts,
	TStaticArray<uint32, EMemoryType::GPU> Dag,
	uint32 Index)
{
	// No need to be thread safe here, worst case we do the same computation twice
	if (Counts[Index] != 0)
	{
		return Counts[Index];
	}

	const uint32 ChildMask = Utils::ChildMask(Dag[Index]);
	uint64 Count = 0;
	for (uint32 ChildIndex = 0; ChildIndex < Utils::Popc(ChildMask); ChildIndex++)
	{
		Count += SetLeavesCounts<Level + 1>(
			Counts,
			Dag,
			Dag[Index + 1 + ChildIndex]);
	}
	checkInf(Count, (1u << 24));
	Dag[Index] |= Count << 8;
	Counts[Index] = Count;
	return Count;
}

template<>
HOST_DEVICE_RECURSIVE uint64 SetLeavesCounts<LEVELS - 2>(
	TStaticArray<uint64, EMemoryType::GPU> Counts,
	TStaticArray<uint32, EMemoryType::GPU> Dag,
	uint32 Index)
{
	(void)Counts;
	return Utils::Popc(Dag[Index + 0]) + Utils::Popc(Dag[Index + 1]);
}

__global__ void SetLeavesCountsKernel(
	TStaticArray<uint32, EMemoryType::GPU> Indices,
	TStaticArray<uint64, EMemoryType::GPU> Counts,
	TStaticArray<uint32, EMemoryType::GPU> Dag)
{
	const uint32 Index = blockIdx.x * blockDim.x + threadIdx.x;
	if (!Indices.IsValidIndex(Index)) return;

	const uint64 Count = SetLeavesCounts<TOP_LEVEL>(Counts, Dag, Indices[Index]);
	checkEqual(Count, Counts[Indices[Index]]);
}

template<bool IsFirstPass>
uint64 SetLeavesCountsCpuImpl(
	TStaticArray<uint32, EMemoryType::CPU> Dag,
	TStaticArray<uint64, EMemoryType::CPU> Counts,
	std::vector<uint64>& OutEnclosedLeaves,
	std::vector<uint32>& OutIndices,
	uint32 Index,
	uint32 Level)
{
	if (!IsFirstPass && Counts[Index] != 0)
	{
		return Counts[Index];
	}

	if (Level == TOP_LEVEL)
	{
		// Must have Counts[Index] != 0 in second pass
		check(IsFirstPass);
		OutIndices.push_back(Index);
		return 0;
	}
	else 
	{
		const uint32 ChildMask = Utils::ChildMask(Dag[Index]);
		uint64 Count = 0;
		for (uint32 ChildIndex = 0; ChildIndex < Utils::Popc(ChildMask); ChildIndex++)
		{
			Count += SetLeavesCountsCpuImpl<IsFirstPass>(
				Dag,
				Counts,
				OutEnclosedLeaves,
				OutIndices,
				Dag[Index + 1 + ChildIndex],
				Level + 1);
		}
		if (!IsFirstPass)
		{
			const uint64 EnclosedLeavesIndex = OutEnclosedLeaves.size();
			checkInf(EnclosedLeavesIndex, 1u << 24);
			checkEqual(Dag[Index] >> 8, 0);
			Dag[Index] |= EnclosedLeavesIndex << 8;
			OutEnclosedLeaves.push_back(Count);
			Counts[Index] = Count;
		}
		return Count;
	}
}

void SetLeavesCountsCpu(FFinalDag& Dag)
{
	PROFILE_FUNCTION();

	std::vector<uint64> EnclosedLeaves;
	std::vector<uint32> Indices;

	auto GpuDag = Dag.Dag;
	auto CpuDag = GpuDag.CreateCPU();
	TStaticArray<uint64, EMemoryType::CPU> CpuCounts("Counts", CpuDag.Num());
	CpuCounts.MemSet(0);
	SetLeavesCountsCpuImpl<true>(CpuDag, CpuCounts, EnclosedLeaves, Indices, 0, 0);

	TStaticArray<uint32, EMemoryType::GPU> GpuIndices("Indices", Indices.size());
	{
		PROFILE_SCOPE("Memcpy");
		CUDA_CHECKED_CALL hipMemcpyAsync(GpuIndices.GetData(), &Indices[0], GpuIndices.SizeInBytes(), hipMemcpyHostToDevice);
		CUDA_SYNCHRONIZE_STREAM();
	}

	auto GpuCounts = CpuCounts.CreateGPU();
	SetLeavesCountsKernel <<< uint32(Indices.size()), 1 >>> (GpuIndices, GpuCounts, GpuDag);
	GpuIndices.Free();

	GpuCounts.CopyToCPU(CpuCounts);
	GpuCounts.Free();

	GpuDag.CopyToCPU(CpuDag);

	SetLeavesCountsCpuImpl<false>(CpuDag, CpuCounts, EnclosedLeaves, Indices, 0, 0);
	CpuCounts.Free();
	
	CpuDag.CopyToGPU(Dag.Dag);
	CpuDag.Free();

	if (!EnclosedLeaves.empty())
	{
		Dag.EnclosedLeaves = TStaticArray<uint64, EMemoryType::GPU>("EnclosedLeaves", EnclosedLeaves.size());
		{
			PROFILE_SCOPE("Memcpy");
			CUDA_CHECKED_CALL hipMemcpyAsync(Dag.EnclosedLeaves.GetData(), &EnclosedLeaves[0], Dag.EnclosedLeaves.SizeInBytes(), hipMemcpyHostToDevice);
			CUDA_SYNCHRONIZE_STREAM();
		}
	}
}

///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////

FFinalDag DAGCompression::CreateFinalDAG(FCpuDag&& CpuDag)
{
	PROFILE_FUNCTION();
	
	std::vector<FGpuLevel> Levels;
	for (auto& Level : CpuDag.Levels)
	{
		Levels.push_back({});
		Levels.back().FromCPU(Level, false);
		Level.Free();
	}
	auto Leaves = CpuDag.Leaves.CreateGPU();
	CpuDag.Leaves.Free();

	struct FLevelInfo
	{
		uint32 NumWords;
		TStaticArray<uint32, EMemoryType::GPU> ChildPositions;
	};

	std::vector<FLevelInfo> LevelsInfo;
	LevelsInfo.reserve(Levels.size());
	
	uint64 Num = 0;
	for (auto& Level : Levels)
	{
		FLevelInfo Info;
		Info.NumWords = ComputeChildPositions(Level, Info.ChildPositions);
		Num += Info.NumWords;
		LevelsInfo.push_back(Info);
	}
	Num += Leaves.Num() * 2;

	TStaticArray<uint32, EMemoryType::GPU> Dag("Dag", Num);
	Dag.MemSet(0xFF); // To spot errors
	uint64 Offset = 0;
	for (uint64 LevelIndex = 0; LevelIndex < Levels.size(); LevelIndex++)
	{
		const auto& Level = Levels[LevelIndex];
		const auto& LevelInfo = LevelsInfo[LevelIndex];
		const uint32 NextLevelStart = Cast<uint32>(Offset + LevelInfo.NumWords);
		checkInfEqual(Offset, Num);
		auto DagWithOffset = TStaticArray<uint32, EMemoryType::GPU>(Dag.GetData() + Offset, Num - Offset);
		if (LevelIndex == Levels.size() - 1)
		{
			const auto GetChildIndex = [=] GPU_LAMBDA (uint32 Index) { return NextLevelStart + 2 * Index; };
			WriteLevelTo(Level, LevelInfo.ChildPositions, DagWithOffset, GetChildIndex);
		}
		else
		{
			const auto NextLevelChildPositions = LevelsInfo[LevelIndex + 1].ChildPositions;
			const auto GetChildIndex = [=] GPU_LAMBDA (uint32 Index) { return NextLevelStart + NextLevelChildPositions[Index]; };
			WriteLevelTo(Level, LevelInfo.ChildPositions, DagWithOffset, GetChildIndex);
		}
		Offset += LevelInfo.NumWords;
	}
	checkEqual(Offset + 2 * Leaves.Num(), Num);
	CUDA_CHECKED_CALL hipMemcpyAsync(Dag.GetData() + Offset, Leaves.GetData(), Leaves.Num() * sizeof(uint64), hipMemcpyDeviceToDevice);
	CUDA_SYNCHRONIZE_STREAM();

	for (auto& Level : Levels)
	{
		Level.Free(false);
	}	
	for (auto& LevelInfo : LevelsInfo)
	{
		LevelInfo.ChildPositions.Free();
	}
	Leaves.Free();

	FFinalDag FinalDag;
	FinalDag.Dag = Dag;

#if ENABLE_COLORS
	SetLeavesCountsCpu(FinalDag);
	{
		PROFILE_SCOPE("Copy Colors");
		FinalDag.Colors = TStaticArray<uint32, EMemoryType::GPU>("Colors", CpuDag.Colors.Num());
		CpuDag.Colors.CopyToGPU(FinalDag.Colors);
		CpuDag.Colors.Free();
	}
#endif

	CheckDag(FinalDag);
	return FinalDag;
}