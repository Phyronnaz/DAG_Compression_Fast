#include "hip/hip_runtime.h"
#include "DAGCompression/DAGCompression.h"
#include "GpuPrimitives.h"

HOST_DEVICE uint64 HashChildren(const FChildrenIndices& ChildrenIndices, const TGpuArray<uint64>& ChildrenArray)
{
	uint64 ChildrenHashes[8];
	for (int32 Index = 0; Index < 8; Index++)
	{
		const uint32 ChildIndex = ChildrenIndices.Indices[Index];
		ChildrenHashes[Index] = ChildIndex == 0xFFFFFFFF ? 0 : ChildrenArray[ChildIndex];
	}
	return HashChildrenHashes(ChildrenHashes);
}

void DAGCompression::ComputeHashes(FGpuLevel& Level, const TGpuArray<uint64>& LowerLevelHashes)
{
	Level.Hashes = TGpuArray<uint64>("Hashes", Level.ChildrenIndices.Num());
	Transform(Level.ChildrenIndices, Level.Hashes, [=] GPU_LAMBDA (const FChildrenIndices & Children) { return HashChildren(Children, LowerLevelHashes); });

#if ENABLE_CHECKS && DEBUG_GPU_ARRAYS
	std::unordered_map<uint64, FChildrenIndices> HashesToIndices;
	for (uint64 Index = 0; Index < Level.Hashes.Num(); Index++)
	{
		const uint64 Hash = Level.Hashes[Index];
		const FChildrenIndices ChildrenIndices = Level.ChildrenIndices[Index];

		const auto ExistingIndices = HashesToIndices.find(Hash);
		if (ExistingIndices == HashesToIndices.end())
		{
			HashesToIndices[Hash] = ChildrenIndices;
		}
		else
		{
			for (int32 ChildIndex = 0; ChildIndex < 8; ChildIndex++)
			{
				checkEqual(ChildrenIndices.Indices[ChildIndex], ExistingIndices->second.Indices[ChildIndex]);
			}
		}
	}
#endif
}

TGpuArray<uint64> DAGCompression::SortFragmentsAndRemoveDuplicates(TGpuArray<uint64> Fragments)
{
	PROFILE_FUNCTION();

	auto NewFragments = TGpuArray<uint64>("Fragments", Fragments.Num());

	{
		const int32 Num = Cast<int32>(Fragments.Num());
		const int32 NumBits = 3 * SUBDAG_LEVELS;
		hipcub::DoubleBuffer<uint64> Keys(Fragments.GetData(), NewFragments.GetData());
		
		void* TempStorage = nullptr;
		size_t TempStorageBytes;
		CUDA_CHECKED_CALL hipcub::DeviceRadixSort::SortKeys(TempStorage, TempStorageBytes, Keys, Num, 0, NumBits);
		CUDA_CHECKED_CALL cnmemMalloc(&TempStorage, TempStorageBytes, DEFAULT_STREAM);
		CUDA_CHECKED_CALL hipcub::DeviceRadixSort::SortKeys(TempStorage, TempStorageBytes, Keys, Num, 0, NumBits);
		CUDA_CHECK_ERROR();
		CUDA_CHECKED_CALL cnmemFree(TempStorage, DEFAULT_STREAM);

		if (Keys.Current() != NewFragments.GetData())
		{
			check(Keys.Current() == Fragments.GetData());
			std::swap(Fragments, NewFragments);
		}
	}

	// Sorted data is in NewFragments

	int32 NumUnique;
	{
		// Don't check the color bits
		const auto EqualityLambda = [] GPU_LAMBDA (uint64 A, uint64 B) { return (A << (64 - 3 * SUBDAG_LEVELS)) == (B << (64 - 3 * SUBDAG_LEVELS)); };
		const int32 Num = Cast<int32>(Fragments.Num());
		TSingleGPUElement<int32> NumUniqueGPU;
		
		void* TempStorage = nullptr;
		size_t TempStorageBytes;
		CUDA_CHECKED_CALL hipcub::DeviceSelect2::Unique(TempStorage, TempStorageBytes, NewFragments.GetData(), Fragments.GetData(), NumUniqueGPU.GetPtr(), Num, EqualityLambda);
		CUDA_CHECKED_CALL cnmemMalloc(&TempStorage, TempStorageBytes, DEFAULT_STREAM);
		CUDA_CHECKED_CALL hipcub::DeviceSelect2::Unique(TempStorage, TempStorageBytes, NewFragments.GetData(), Fragments.GetData(), NumUniqueGPU.GetPtr(), Num, EqualityLambda);
		CUDA_CHECK_ERROR();
		CUDA_CHECKED_CALL cnmemFree(TempStorage, DEFAULT_STREAM);

		NumUnique = NumUniqueGPU.GetValue();
	}
	checkInf(NumUnique, Fragments.Num());

	// Unique data is in Fragments

	// Shrink
	NewFragments.Free();
	NewFragments = TGpuArray<uint64>("Fragments", NumUnique);
	CUDA_CHECKED_CALL hipMemcpyAsync(NewFragments.GetData(), Fragments.GetData(), NumUnique * sizeof(uint64), hipMemcpyDeviceToDevice);
	CUDA_SYNCHRONIZE_STREAM();

	Fragments.Free();
	
	return NewFragments;
}

TGpuArray<uint32> DAGCompression::ExtractColorsAndFixFragments(TGpuArray<uint64>& Fragments)
{
	PROFILE_FUNCTION();

	auto Colors = TGpuArray<uint32>("Colors", Fragments.Num());
	Transform(Fragments, Colors, [] GPU_LAMBDA (uint64 X) { return X >> 40; });
	Transform(Fragments, Fragments, [] GPU_LAMBDA (uint64 X) { return X & ((uint64(1) << 40) - 1); });
	return Colors;
}

TGpuArray<uint64> DAGCompression::ExtractLeavesAndShiftReduceFragments(TGpuArray<uint64>& Fragments)
{
	PROFILE_FUNCTION();

	// Use the last 6 bits to create the leaves
	const auto ParentsTransform = [] GPU_LAMBDA (uint64 Code) { return Code >> 6; };
	const auto LeavesTransform = [] GPU_LAMBDA (uint64 Code) { return uint64(1) << (Code & 0b111111); };

	const int32 MinAverageVoxelsPer64Leaf = 1; // TODO higher?
	// TODO compute exact num?
	
	auto NewFragments = TGpuArray<uint64>("Fragments", Fragments.Num() / MinAverageVoxelsPer64Leaf);
	auto Leaves = TGpuArray<uint64>("Leaves", Fragments.Num() / MinAverageVoxelsPer64Leaf);
	
	const auto ParentsBits = TransformIterator(Fragments.GetData(), ParentsTransform);
	const auto LeavesBits = TransformIterator(Fragments.GetData(), LeavesTransform);

	const int32 Num = Cast<int32>(Fragments.Num());

	TSingleGPUElement<int32> NumRunsGPU;
	{
		void* TempStorage = nullptr;
		size_t TempStorageBytes = 0;
		CUDA_CHECKED_CALL hipcub::DeviceReduce::ReduceByKey(TempStorage, TempStorageBytes, ParentsBits, NewFragments.GetData(), LeavesBits, Leaves.GetData(), NumRunsGPU.GetPtr(), thrust::bit_or<uint64>(), Num);
		CUDA_CHECKED_CALL cnmemMalloc(&TempStorage, TempStorageBytes, DEFAULT_STREAM);
		CUDA_CHECKED_CALL hipcub::DeviceReduce::ReduceByKey(TempStorage, TempStorageBytes, ParentsBits, NewFragments.GetData(), LeavesBits, Leaves.GetData(), NumRunsGPU.GetPtr(), thrust::bit_or<uint64>(), Num);
		CUDA_CHECK_ERROR();
		CUDA_CHECKED_CALL cnmemFree(TempStorage, DEFAULT_STREAM);
	}
	
	const int32 NumRuns = NumRunsGPU.GetValue();
	checkfAlways(NumRuns <= NewFragments.Num(), "Need to decrease MinAverageVoxelsPer64Leaf: %d runs, but expected max %d", NumRuns, int32(NewFragments.Num()));

	// Note: these copies & allocs could be avoided if we computed the exact number of runs first

	Fragments.Free();
	Fragments = TGpuArray<uint64>("Fragments", NumRuns);
	CUDA_CHECKED_CALL hipMemcpyAsync(Fragments.GetData(), NewFragments.GetData(), NumRuns * sizeof(uint64), hipMemcpyDeviceToDevice);
	CUDA_SYNCHRONIZE_STREAM();
	NewFragments.Free();

	auto ShrunkLeaves = TGpuArray<uint64>("Leaves", NumRuns);
	CUDA_CHECKED_CALL hipMemcpyAsync(ShrunkLeaves.GetData(), Leaves.GetData(), NumRuns * sizeof(uint64), hipMemcpyDeviceToDevice);
	CUDA_SYNCHRONIZE_STREAM();
	Leaves.Free();

	return ShrunkLeaves;
}

void SortLevelImpl(bool IsLeaf, FGpuLevel& Level, TGpuArray<uint32>& OutHashesToSortedUniqueHashes)
{
	PROFILE_FUNCTION();

	using namespace DAGCompression;
	
	CheckLevelIndices(Level);

	const int32 NumHashes = Cast<int32>(Level.Hashes.Num());
	auto SortedHashes = TGpuArray<uint64>("SortedHashes", NumHashes);
	auto SortedHashesToHashes = TGpuArray<uint32>("SortedHashesToHashes", NumHashes);

	{
		// Need a real array to use CUB sort
		auto Sequence = TGpuArray<uint32>("Sequence", NumHashes);
		MakeSequence(Sequence);

		hipcub::DoubleBuffer<uint64> Keys(Level.Hashes.GetData(), SortedHashes.GetData());
		hipcub::DoubleBuffer<uint32> Values(Sequence.GetData(), SortedHashesToHashes.GetData());

		void* TempStorage = nullptr;
		size_t TempStorageBytes;
		CUDA_CHECKED_CALL hipcub::DeviceRadixSort::SortPairs(TempStorage, TempStorageBytes, Keys, Values, NumHashes, 0, 64);
		CUDA_CHECKED_CALL cnmemMalloc(&TempStorage, TempStorageBytes, DEFAULT_STREAM);
		CUDA_CHECKED_CALL hipcub::DeviceRadixSort::SortPairs(TempStorage, TempStorageBytes, Keys, Values, NumHashes, 0, 64);
		CUDA_CHECK_ERROR();
		CUDA_CHECKED_CALL cnmemFree(TempStorage, DEFAULT_STREAM);

		if (Keys.Current() != SortedHashes.GetData())
		{
			check(Keys.Current() == Level.Hashes.GetData());
			std::swap(SortedHashes, Level.Hashes);
		}
		if (Values.Current() != SortedHashesToHashes.GetData())
		{
			check(Values.Current() == Sequence.GetData());
			std::swap(SortedHashesToHashes, Sequence);
		}

		// Level hashes are trashed by the sort
		Level.Hashes.Free();
		Sequence.Free();
	}
	CheckIsSorted(SortedHashes);

	auto SortedHashesFlags = TGpuArray<uint32>("SortedHashesFlags", NumHashes);
	AdjacentDifference(SortedHashes, SortedHashesFlags, thrust::not_equal_to<uint64>(), 0u);

	auto SortedHashesToUniqueHashes = TGpuArray<uint32>("SortedHashesToUniqueHashes", NumHashes);
	{
		void* TempStorage = nullptr;
		size_t TempStorageBytes;
		CUDA_CHECKED_CALL hipcub::DeviceScan::InclusiveSum(TempStorage, TempStorageBytes, SortedHashesFlags.GetData(), SortedHashesToUniqueHashes.GetData(), NumHashes);
		CUDA_CHECKED_CALL cnmemMalloc(&TempStorage, TempStorageBytes, DEFAULT_STREAM);
		CUDA_CHECKED_CALL hipcub::DeviceScan::InclusiveSum(TempStorage, TempStorageBytes, SortedHashesFlags.GetData(), SortedHashesToUniqueHashes.GetData(), NumHashes);
		CUDA_CHECK_ERROR();
		CUDA_CHECKED_CALL cnmemFree(TempStorage, DEFAULT_STREAM);
	}
	SortedHashesFlags.Free();
	
	const uint32 NumUniques = GetElement(SortedHashesToUniqueHashes, SortedHashesToUniqueHashes.Num() - 1) + 1;

	Level.Hashes = TGpuArray<uint64>(IsLeaf ? "Leaves" : "Hashes", NumUniques);
	Scatter(SortedHashes, SortedHashesToUniqueHashes, Level.Hashes);
	SortedHashes.Free();
	CheckIsSorted(Level.Hashes);

	OutHashesToSortedUniqueHashes = TGpuArray<uint32>("HashesToSortedUniqueHashes", NumHashes);
	Scatter(SortedHashesToUniqueHashes, SortedHashesToHashes, OutHashesToSortedUniqueHashes);
	CheckArrayBounds(OutHashesToSortedUniqueHashes, 0u, NumUniques - 1);
	SortedHashesToHashes.Free();
	SortedHashesToUniqueHashes.Free();

	if (!IsLeaf)
	{
		{
			auto NewChildrenIndices = TGpuArray<FChildrenIndices>("ChildrenIndices", NumUniques);
			Scatter(Level.ChildrenIndices, OutHashesToSortedUniqueHashes, NewChildrenIndices);
			Level.ChildrenIndices.Free();
			Level.ChildrenIndices = NewChildrenIndices;
		}
		{
			auto NewChildMasks = TGpuArray<uint8>("ChildMasks", NumUniques);
			Scatter(Level.ChildMasks, OutHashesToSortedUniqueHashes, NewChildMasks);
			Level.ChildMasks.Free();
			Level.ChildMasks = NewChildMasks;
		}
	}

	CheckLevelIndices(Level);
}

void DAGCompression::SortLeaves(TGpuArray<uint64>& Leaves, TGpuArray<uint32>& OutHashesToSortedUniqueHashes)
{
	FGpuLevel Level;
	Level.Hashes = std::move(Leaves);
	SortLevelImpl(true, Level, OutHashesToSortedUniqueHashes);
	Leaves = Level.Hashes;
}

void DAGCompression::SortLevel(FGpuLevel& Level, TGpuArray<uint32>& OutHashesToSortedUniqueHashes)
{
	SortLevelImpl(false, Level, OutHashesToSortedUniqueHashes);
}

FGpuLevel DAGCompression::ExtractLevelAndShiftReduceFragments(TGpuArray<uint64>& Fragments, const TGpuArray<uint32>& FragmentIndicesToChildrenIndices)
{
	PROFILE_FUNCTION();

	FGpuLevel OutLevel;

	const int32 Num = Cast<int32>(Fragments.Num());

	const auto ParentsTransform = [] GPU_LAMBDA(uint64 Code) { return Code >> 3; };
	const auto ChildrenTransform = [] GPU_LAMBDA(uint64 Code) { return uint64(1) << (Code & 0b111); };

	// First create the mapping & get the number of unique parents
	int32 NumUniqueParents;
	{
		auto UniqueParentsSum = TGpuArray<uint32>("UniqueParentsSum", Num);
		{
			auto UniqueParentsFlags = TGpuArray<uint32>("UniqueParentsFlags", Num);
			AdjacentDifferenceWithTransform(Fragments, ParentsTransform, UniqueParentsFlags, thrust::not_equal_to<uint64>(), 0u);
			{
				void* TempStorage = nullptr;
				size_t TempStorageBytes;
				CUDA_CHECKED_CALL hipcub::DeviceScan::InclusiveSum(TempStorage, TempStorageBytes, UniqueParentsFlags.GetData(), UniqueParentsSum.GetData(), Num);
				CUDA_CHECKED_CALL cnmemMalloc(&TempStorage, TempStorageBytes, DEFAULT_STREAM);
				CUDA_CHECKED_CALL hipcub::DeviceScan::InclusiveSum(TempStorage, TempStorageBytes, UniqueParentsFlags.GetData(), UniqueParentsSum.GetData(), Num);
				CUDA_CHECK_ERROR();
				CUDA_CHECKED_CALL cnmemFree(TempStorage, DEFAULT_STREAM);
			}
			UniqueParentsFlags.Free();
		}

		NumUniqueParents = GetElement(UniqueParentsSum, UniqueParentsSum.Num() - 1) + 1;
		checkInfEqual(NumUniqueParents, Num);

		OutLevel.ChildrenIndices = TGpuArray<FChildrenIndices>("ChildrenIndices", NumUniqueParents);
		OutLevel.ChildrenIndices.MemSet(0xFF);
		{
			const auto CompactChildrenToExpandedChildren = [=] GPU_LAMBDA(uint64 Index) { return (Fragments[Index] & 0b111) + 8 * UniqueParentsSum[Index]; };
			auto Out = OutLevel.ChildrenIndices.CastTo<uint32>();
			ScatterPred(FragmentIndicesToChildrenIndices, CompactChildrenToExpandedChildren, Out);
		}

		UniqueParentsSum.Free();
	}

	auto NewFragments = TGpuArray<uint64>("Fragments", NumUniqueParents);
	OutLevel.ChildMasks = TGpuArray<uint8>("ChildMasks", NumUniqueParents);
	{
		const auto ParentsBits = TransformIterator(Fragments.GetData(), ParentsTransform);
		const auto ChildrenBits = TransformIterator(Fragments.GetData(), ChildrenTransform);

		TSingleGPUElement<int32> NumRunsGPU;

		void* TempStorage = nullptr;
		size_t TempStorageBytes = 0;
		CUDA_CHECKED_CALL hipcub::DeviceReduce::ReduceByKey(TempStorage, TempStorageBytes, ParentsBits, NewFragments.GetData(), ChildrenBits, OutLevel.ChildMasks.GetData(), NumRunsGPU.GetPtr(), thrust::bit_or<uint8>(), Num);
		CUDA_CHECKED_CALL cnmemMalloc(&TempStorage, TempStorageBytes, DEFAULT_STREAM);
		CUDA_CHECKED_CALL hipcub::DeviceReduce::ReduceByKey(TempStorage, TempStorageBytes, ParentsBits, NewFragments.GetData(), ChildrenBits, OutLevel.ChildMasks.GetData(), NumRunsGPU.GetPtr(), thrust::bit_or<uint8>(), Num);
		CUDA_CHECK_ERROR();
		CUDA_CHECKED_CALL cnmemFree(TempStorage, DEFAULT_STREAM);

		checkEqual(NumUniqueParents, NumRunsGPU.GetValue());
	}

	Fragments.Free();
	Fragments = NewFragments;

	CheckLevelIndices(OutLevel);
	return OutLevel;
}

FCpuDag DAGCompression::CreateSubDAG(TGpuArray<uint64>& InFragments)
{
	PROFILE_FUNCTION();
	
	FCpuDag CpuDag;
	CpuDag.Levels.resize(SUBDAG_LEVELS - 2);

	if (InFragments.Num() == 0)
	{
		return CpuDag;
	}

	TGpuArray<uint64> Fragments;

	{
		auto InFragmentsCopy = TGpuArray<uint64>("FragmentsCopy", InFragments.Num());
		CUDA_CHECKED_CALL hipMemcpyAsync(InFragmentsCopy.GetData(), InFragments.GetData(), InFragments.SizeInBytes(), hipMemcpyDeviceToDevice);
		CUDA_SYNCHRONIZE_STREAM();
		Fragments = SortFragmentsAndRemoveDuplicates(std::move(InFragmentsCopy));
	}
	
#if ENABLE_COLORS
	auto Colors = ExtractColorsAndFixFragments(Fragments);
	CpuDag.Colors = Colors.CreateCPU();
	Colors.Free();
#endif

	auto Leaves = ExtractLeavesAndShiftReduceFragments(Fragments);

	TGpuArray<uint32> FragmentIndicesToChildrenIndices;
	SortLeaves(Leaves, FragmentIndicesToChildrenIndices);
	CpuDag.Leaves = Leaves.CreateCPU();
	
	auto PreviousLevelHashes = Leaves;
	for (int32 LevelDepth = SUBDAG_LEVELS - 3; LevelDepth >= 0; LevelDepth--)
	{
		PROFILE_SCOPE("Level %d", LevelDepth);
		LOG_DEBUG("Level %d", LevelDepth);
		
		FGpuLevel Level = ExtractLevelAndShiftReduceFragments(Fragments, FragmentIndicesToChildrenIndices);
		FragmentIndicesToChildrenIndices.Free();

		ComputeHashes(Level, PreviousLevelHashes);
		PreviousLevelHashes.Free();

		SortLevel(Level, FragmentIndicesToChildrenIndices);

		CpuDag.Levels[LevelDepth] = Level.ToCPU();
		Level.Free(false);
		PreviousLevelHashes = Level.Hashes;
	}
	check(PreviousLevelHashes.Num() == 1);
	PreviousLevelHashes.Free();

	FragmentIndicesToChildrenIndices.Free();
	Fragments.Free();
	
	return CpuDag;
}